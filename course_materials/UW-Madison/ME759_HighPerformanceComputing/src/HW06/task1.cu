#include "mmul.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>

void assign_one(float *mat, unsigned int n) {
  for (unsigned int i = 0; i < n * n; ++i) {
    mat[i] = 1;
  }
}

int main(int argc, char *argv[]) {
  int n = atoi(argv[1]);
  int n_ntests = atoi(argv[2]);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  assign_one(A, n);
  assign_one(B, n);
  assign_one(C, n);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float ms;
  float ms_sum = 0;
  float ms_avg;

  for (unsigned int i = 0; i < n_ntests; ++i) {
    hipEventRecord(start);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    mmul(handle, A, B, C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ms, start, stop);
    ms_sum += ms;
  }
  ms_avg = (float)(ms_sum / n_ntests);

  // std::cout << C[n * n - 1] << '\n';
  std::cout << ms_avg << '\n';

  hipblasDestroy(handle);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  return 0;
}
