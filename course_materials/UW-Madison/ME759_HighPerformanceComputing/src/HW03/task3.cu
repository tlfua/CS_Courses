#include "vadd.cuh"
#include <cstdio>
#include <cstdlib>
#include <iostream>

#define THREAD_NUM 512
// #define THREAD_NUM 1024

int main(int argc, char *argv[]) {
  int N = atoi(argv[1]);
  float *a, *b;
  hipMallocManaged(&a, N * sizeof(float));
  hipMallocManaged(&b, N * sizeof(float));

  // assign a and b
  for (int i = 0; i < N; i++) {
    a[i] = (float)i;
    b[i] = (float)i;
  }

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  vadd<<<N / THREAD_NUM + 1, THREAD_NUM>>>(a, b, N);
  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // for (int i = 0; i < N; i++) {
  //     std::cout << b[i] << " ";
  // }

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  std::cout << ms / 1000 << "\n";
  std::cout << b[0] << "\n";
  std::cout << b[N - 1] << "\n";

  hipFree(a);
  hipFree(b);
  return 0;
}
