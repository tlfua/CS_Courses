#include "matmul.cuh"
#include <cstdio>
#include <cstdlib>
#include <iostream>

int main(int argc, char *argv[]) {
  int n = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  // sample input
  // A[0] = 1.0;
  // A[1] = 2.0;
  // A[2] = 3.0;
  // A[3] = 4.0;

  // B[0] = 4.0;
  // B[1] = 3.0;
  // B[2] = 2.0;
  // B[3] = 1.0;

  // general input
  for (int i = 0; i < n * n; ++i) {
    A[i] = 1.0;
    B[i] = 1.0;
  }

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  matmul(A, B, C, n, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // sample print
  // for (int i = 0; i < n * n; ++i) {
  //     std::cout << C[i] << " ";
  // }

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  std::cout << C[n * n - 1] << '\n';
  std::cout << ms << '\n';

  hipFree(A);
  hipFree(B);
  hipFree(C);
  return 0;
}
