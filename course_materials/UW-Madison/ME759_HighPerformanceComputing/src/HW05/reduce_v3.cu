#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <iostream>

__global__ void reduce_kernel(const int* g_idata, int* g_odata, unsigned int n)
{ 
    __shared__ unsigned int threads_per_block;
    if (threadIdx.x == 0) {
        threads_per_block = blockDim.x;
    }
    __syncthreads();
    
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = threads_per_block * bid + tid;

    // specially deal with last block
    if ((threadIdx.x == 0) && (blockIdx.x == gridDim.x - 1)) {
        unsigned int last_threads_per_block = n % threads_per_block;
        if (last_threads_per_block != 0) {
            threads_per_block = last_threads_per_block;
        }
    }
    __syncthreads();
    
    extern __shared__ int sh_data[];
    if (tid < threads_per_block) {
        sh_data[tid] = g_idata[global_tid];
    }
    __syncthreads();
    
    // do reduction in shared mem
    /*
    for(unsigned int s = (threads_per_block + 1) / 2; s > 0; s >>= 1) {
        if ((tid < s) && (tid + s < threads_per_block)) {
            sh_data[tid] += sh_data[tid + s];
        }
        __syncthreads();
    }
    */

    unsigned int s;
    unsigned int block_size = threads_per_block;
    while (block_size > 1) {
    
        if ((block_size % 2) == 0) {
	    s = block_size / 2;
	    if (tid < s) {
	        sh_data[tid] += sh_data[tid + s];
	    }
	} else {
	    s = (block_size + 1) / 2;
	    if (tid < s - 1) {
	        sh_data[tid] += sh_data[tid + s];
	    }
	}
	block_size = s;
	__syncthreads();
    }

    /*
    if ((tid == 0) && (bid == gridDim.x - 1)) {
        printf("sh_data:\n");
	for (unsigned int i = 0; i < n; ++i) {
	    printf("%d ", sh_data[i]);
	}
	printf("\n");
    }
    __syncthreads();
    */
    
    // write result for this block to global memory
    if (tid == 0) {
        g_odata[bid] = sh_data[0];
    }
}

__host__ int reduce(const int* arr, unsigned int N, unsigned int threads_per_block)
{
    // arr_size: 32 -> 16 -> 8 -> 4 -> 2 -> 1

    int *idata, *odata;
    hipMallocManaged(&idata, N * sizeof(int));
    hipMemcpy(idata, arr, N * sizeof(int), hipMemcpyHostToDevice);

    hipMallocManaged(&odata, N * sizeof(int));

    // unsigned int idata_size = N;
    unsigned int blocks;
    while (N > 1) {

        blocks = (N + threads_per_block - 1) / threads_per_block;
        reduce_kernel<<<blocks, threads_per_block, threads_per_block * sizeof(int)>>>(idata, odata, N);
        hipDeviceSynchronize();
    
        /*
	    std::cout << "odata:\n";
	    for (unsigned int i = 0; i < blocks; ++i) {
		    std::cout << odata[i] << ' ';
	    }
        std::cout << '\n';
        */
        
        hipMemcpy(idata, odata, blocks * sizeof(int), hipMemcpyDeviceToDevice);
	    N = blocks;
    }

    int res = odata[0];
    hipFree(idata);
    hipFree(odata);
    return res;
}
