#include "matmul.cuh"
#include <cstdio>
#include <cstdlib>
#include <iostream>

int main(int argc, char *argv[]) {

  int n = atoi(argv[1]);
  int block_dim = atoi(argv[2]);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  // general input
  for (int i = 0; i < n * n; ++i) {
    A[i] = 1.0;
    B[i] = 1.0;
  }

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  matmul(A, B, C, n, block_dim);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // sample print
  // for (int i = 0; i < n * n; ++i) {
  //     std::cout << C[i] << " ";
  // }

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  std::cout << C[0] << '\n';
  std::cout << C[n * n - 1] << '\n';
  std::cout << ms << '\n';

  hipFree(A);
  hipFree(B);
  hipFree(C);
  return 0;
}
